#include "hip/hip_runtime.h"
// ///////////////////////////////////////////////////////////////////////////////////////
// LTB Distance Volume Hierarchy
// Copyright (c) 2020 Logan Barnes - All Rights Reserved
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
// ///////////////////////////////////////////////////////////////////////////////////////
#include "dvh_renderable.hpp"

// project
#include "buffer_map_guard.hpp"
#include "ltb/gvs/display/gui/imgui_colors.hpp"

// external
#include <Magnum/GL/Renderer.h>
#include <Magnum/Math/Vector2.h>
#include <glm/ext/scalar_constants.hpp>
#include <glm/gtx/hash.hpp>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

// standard
#include <algorithm>
#include <random>
#include <unordered_set>

//#define NOT_UNIQUE

using namespace Magnum;
using namespace Math::Literals;

namespace ltb {
namespace example {

namespace {

struct DistanceFromCameraComparator {
    LTB_CUDA_FUNC bool operator()(const Cell& c1, const Cell& c2) {
        return glm::distance(camera_position, c1.center_point(level_0_resolution))
            > glm::distance(camera_position, c2.center_point(level_0_resolution));
    }

    glm::vec3 camera_position;
    float     level_0_resolution;
};

//auto to_1d_index(glm::ivec2 const& index, glm::ivec2 const& dimensions) -> std::size_t {
//    return static_cast<std::size_t>(index.y) * static_cast<std::size_t>(dimensions.x) + index.x;
//}
//
//auto to_1d_index(glm::ivec3 const& index, glm::ivec3 const& min_index, glm::ivec3 const& max_index) -> std::size_t {
//    return index.z
//}

auto create_cells() {
    constexpr auto index_radius = 10.f;
    constexpr auto num_points   = 1000000u;

    // Create a bunch of uniformly distributed points on a sphere
#ifdef NOT_UNIQUE
    std::vector<glm::ivec3> points;
    points.reserve(num_points);
#else
    std::unordered_set<glm::ivec3> points;
#endif
    {
        float                                 u, theta, coeff;
        std::mt19937                          gen{std::random_device{}()};
        std::uniform_real_distribution<float> u_dist(-1.f, 1.f);
        std::uniform_real_distribution<float> theta_dist(0.f, 2.f * glm::pi<float>());

        for (auto i = 0u; i < num_points; ++i) {
            // Uniform surface distribution
            u     = u_dist(gen);
            theta = theta_dist(gen);
            coeff = std::sqrt(1.f - u * u);

            auto p = glm::vec3(coeff * std::cos(theta), coeff * std::sin(theta), u);
#ifdef NOT_UNIQUE
            points.emplace_back(glm::ivec3(glm::round(p * index_radius)));
#else
            points.emplace(glm::ivec3(glm::round(p * index_radius)));
#endif
        }
    }

    std::vector<Cell> cells(points.size());

    std::transform(points.begin(), points.end(), cells.begin(), [](const auto& index) {
        return Cell{index, glm::normalize(glm::vec3(index)), 3};
    });

    return cells;
}

} // namespace

LTB_CUDA_FUNC auto Cell::center_point(float level_0_resolution) const -> glm::vec3 {
    return glm::vec3(index) * level_0_resolution * glm::pow(2.f, static_cast<float>(level));
}

DvhRenderable::DvhRenderable(glm::ivec2 viewport) {
    {
        auto cells     = create_cells();
        interop_cells_ = std::make_unique<cuda::GLBuffer<Cell>>(cells);
    }

    glEnable(GL_PROGRAM_POINT_SIZE);

    mesh_.addVertexBuffer(interop_cells_->gl_buffer(),
                          0,
                          dvh::CellShader::Index(),
                          dvh::CellShader::VectorToClosestPoint(),
                          dvh::CellShader::Level());
    mesh_.setCount(interop_cells_->size());
    mesh_.setPrimitive(GL::MeshPrimitive::Points);

    resize(viewport);
}

void DvhRenderable::update(double /*time_step*/) {
    auto guard = cuda::make_gl_buffer_map_guard(*interop_cells_);

    std::size_t buffer_size;
    auto*       raw_device_ptr = interop_cells_->cuda_buffer(&buffer_size);

    auto device_ptr = thrust::device_pointer_cast(raw_device_ptr);

    thrust::sort(device_ptr,
                 device_ptr + static_cast<decltype(device_ptr)::difference_type>(buffer_size),
                 DistanceFromCameraComparator{camera_position_, base_level_resolution_});
}

void DvhRenderable::render(const gvs::CameraPackage& camera_package) const {
    if (!visible_) {
        return;
    }

    GL::Renderer::enable(GL::Renderer::Feature::Blending);
    GL::Renderer::setDepthMask(false);

    GL::Renderer::setBlendEquation(GL::Renderer::BlendEquation::Add, GL::Renderer::BlendEquation::Add);
    GL::Renderer::setBlendFunction(GL::Renderer::BlendFunction::SourceAlpha,
                                   GL::Renderer::BlendFunction::OneMinusSourceAlpha,
                                   GL::Renderer::BlendFunction::One,
                                   GL::Renderer::BlendFunction::Zero);

    shader_
        .set_projection_from_world_matrix(camera_package.camera->projectionMatrix()
                                          * camera_package.camera->cameraMatrix())
        .set_projection_from_view_matrix(camera_package.camera->projectionMatrix())
        .set_viewport_height(viewport_.y)
        .set_base_level_resolution(base_level_resolution_)
        .set_coloring_type(dvh::CellColoring::UniformColor)
        .set_shading_type(gvs::Shading::Lambertian);

    mesh_.draw(shader_);

    GL::Renderer::setDepthMask(true);
    GL::Renderer::disable(GL::Renderer::Feature::Blending);
}

void DvhRenderable::configure_gui() {
    ImGui::Checkbox("###dvh_visible", &visible_);

    ImGui::SameLine();
    if (ImGui::TreeNode("DVH")) {
        ImGui::TextColored(gvs::gray(), "No settings");
        ImGui::TreePop();
    }
}

void DvhRenderable::resize(glm::ivec2 viewport) {
    viewport_ = viewport;
}
void DvhRenderable::set_camera_position(glm::vec3 cam_pos) {
    camera_position_ = cam_pos;
}

} // namespace example
} // namespace ltb
