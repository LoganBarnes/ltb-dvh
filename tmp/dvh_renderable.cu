#include "hip/hip_runtime.h"
// ///////////////////////////////////////////////////////////////////////////////////////
// LTB Distance Volume Hierarchy
// Copyright (c) 2020 Logan Barnes - All Rights Reserved
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
// ///////////////////////////////////////////////////////////////////////////////////////
#include "dvh_renderable.hpp"

// project
#include "buffer_map_guard.hpp"

// standard
#include <random>

namespace ltb {
namespace example {

namespace {

//// Cuda kernel to test the CUDA side of things
//__global__ void update_particles(gpu::device_buffer<Particle> particles, float timestep) {
//    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if (idx >= particles.length()) {
//        return;
//    }
//
//    // Apply euler update
//    auto& particle = particles[idx];
//    particle.position += particle.velocity * timestep;
//
//    // reverse velocity ("bounce") if we hit a 100x100x100 cube
//    for (int i = 0; i < 3; ++i) {
//        if (fabsf(particle.position[i]) > 50.f) {
//            particle.velocity[i] = -particle.velocity[i];
//        }
//    }
//}

} // namespace

DvhRenderable::DvhRenderable() {
    {
        std::vector<glm::vec4> particles(1000000, glm::vec4(1.f));
        interop_boxes_ = std::make_unique<cuda::GLBuffer<glm::vec4>>(particles);
    }

    mesh_.addVertexBuffer(interop_boxes_->gl_buffer(), 0, gvs::BoxShader::CenterAndRadius());
    mesh_.setCount(interop_boxes_->size());
    mesh_.setPrimitive(Magnum::GL::MeshPrimitive::Points);
}

void DvhRenderable::update(double /*time_step*/) {
    auto guard = cuda::make_gl_buffer_map_guard(*interop_boxes_);
    //    gpu::launch_default(update_particles,
    //                        interop_particles_->size(),
    //                        *interop_particles_,
    //                        static_cast<float>(time_step));
}

void DvhRenderable::render(const gvs::CameraPackage& camera_package) const {
    auto projection_from_world
        = camera_package.camera->projectionMatrix() * camera_package.object.transformationMatrix();

    shader_.set_projection_from_world_matrix(projection_from_world)
        .set_coloring(gvs::Coloring::Normals)
        .set_shading(gvs::Shading::UniformColor);

    mesh_.draw(shader_);
}

void DvhRenderable::configure_gui() {}

} // namespace example
} // namespace ltb
