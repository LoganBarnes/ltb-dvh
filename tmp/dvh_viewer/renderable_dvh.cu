#include "hip/hip_runtime.h"
// ///////////////////////////////////////////////////////////////////////////////////////
// LTB Distance Volume Hierarchy
// Copyright (c) 2020 Logan Barnes - All Rights Reserved
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
// ///////////////////////////////////////////////////////////////////////////////////////
#include "renderable_dvh.hpp"

// project
#include "carpet/generic_guard.h"
#include "carpet/paths.h"
#include "gl/camera/camera.h"
#include "gl/opengl/program.h"
#include "gl/opengl/vertex_array.h"
#include "gl/shaders/gl_shaders.h"
#include "gpu/gl-interop/gl_buffer_map_guard.h"
#include "mesh/primitive_meshes.h"

// external
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

// standard
#include <random>

#ifdef PROPER_CELL
#else
#endif

namespace {

struct DistanceFromCameraComparator {
    CUDA_FUNC bool operator()(const Cell& c1, const Cell& c2) {
#ifdef PROPER_CELL
        return num::distance(camera_position, num::vec3::make_from(c1.index))
            > num::distance(camera_position, num::vec3::make_from(c2.index));
#else
        return num::distance(camera_position, num::vec3::make_from(c1))
            > num::distance(camera_position, num::vec3::make_from(c2));
#endif
    }

    num::vec3 camera_position;
};

} // namespace

/**
 * The particle update and render logic.
 */
RenderableDvh::RenderableDvh(const num::ivec2& viewport) {
    {
        constexpr auto index_radius = 20.f;
        constexpr auto num_points   = 1000000u;

        // Create a bunch of uniformly distributed points on a sphere
        std::vector<num::vec3> points;
        points.reserve(num_points);
        {
            float                                 u, theta, coeff;
            std::mt19937                          gen{std::random_device{}()};
            std::uniform_real_distribution<float> u_dist(-1.f, 1.f);
            std::uniform_real_distribution<float> theta_dist(0.f, 2.f * num::constants::pi<float>());

            for (auto i = 0u; i < num_points; ++i) {
                // Uniform surface distribution
                u     = u_dist(gen);
                theta = theta_dist(gen);
                coeff = std::sqrt(1.f - u * u);

                points.emplace_back(coeff * std::cos(theta), coeff * std::sin(theta), u);
            }
        }

        std::vector<Cell> cells(points.size());

        std::transform(points.begin(), points.end(), cells.begin(), [](const auto& p3) {
#ifdef PROPER_CELL
            return Cell{num::lround<int>(p3 * index_radius), num::normalize(p3), 1.f, 3};
#else
            return num::swizzle(p3 * index_radius, 0.5f);
#endif
        });

        interop_cells_ = std::make_unique<gpu::GLBuffer<Cell>>(cells);
    }

    glEnable(GL_PROGRAM_POINT_SIZE);

    auto shader_path     = carpet::paths::project_root() / "loop" / "src" / "examples" / "dvh_viewer" / "shaders";
    auto shader_path_str = shader_path.string() + "/";

    auto point_shader_vert = gl::ShaderInfo::make_dynamic_shader(
#include "shaders/point_shader.vert"
        , "point_shader.vert", shader_path_str);

    auto point_shader_frag = gl::ShaderInfo::make_dynamic_shader(
#include "shaders/point_shader.frag"
        , "point_shader.frag", shader_path_str);

#ifdef PROPER_CELL
    std::vector<gl::VAOElement> elements{{"index", 3, GL_INT, reinterpret_cast<void*>(offsetof(Cell, index))},
                                         {"vector_to_closest_point",
                                          3,
                                          GL_FLOAT,
                                          reinterpret_cast<void*>(offsetof(Cell, vector_to_closest_point))},
                                         {"distance_to_closest_point",
                                          1,
                                          GL_FLOAT,
                                          reinterpret_cast<void*>(offsetof(Cell, distance_to_closest_point))},
                                         {"level", 1, GL_INT, reinterpret_cast<void*>(offsetof(Cell, level))}};
#else
    std::vector<gl::VAOElement> elements{{"center_and_radius", 4, GL_FLOAT, nullptr}};
#endif

    glpl_.program = std::make_shared<gl::Program>(point_shader_vert, point_shader_frag);
    glpl_.vao = std::make_shared<gl::VertexArray>(*glpl_.program, interop_cells_->gl_buffer(), sizeof(Cell), elements);

    resize(viewport);
}

void RenderableDvh::update(double /*time_step*/) {
    auto guard = gpu::make_gl_buffer_map_guard(*interop_cells_);

    std::size_t buffer_size;
    auto*       raw_device_ptr = interop_cells_->cuda_buffer(&buffer_size);

    auto device_ptr = thrust::device_pointer_cast(raw_device_ptr);

    thrust::sort(device_ptr, device_ptr + buffer_size, DistanceFromCameraComparator{camera_position_});
}

void RenderableDvh::on_render(const gl::Camera& camera,
                              const loop::SceneItem& /*item*/,
                              const loop::OpenGLSceneBackend& /*opengl_backend*/,
                              const glm::mat4& /*full_transform*/) const {

    auto scoped_blend         = carpet::make_guard(glEnable, glDisable, GL_BLEND);
    auto scoped_no_depth_mask = carpet::make_guard([] { glDepthMask(GL_FALSE); }, [] { glDepthMask(GL_TRUE); });

    glBlendEquationSeparate(GL_FUNC_ADD, GL_FUNC_ADD);
    glBlendFuncSeparate(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA, GL_ONE, GL_ZERO);

    // OpenGL rendering to test the OpenGL side of things
    glpl_.program->use([&] {
        glpl_.program->set_uniform(camera.get_projection_view_matrix(), "projection_from_world");
        glpl_.program->set_uniform(camera.get_projection_matrix(), "projection_from_view");
        glpl_.program->set_uniform(viewport_height_, "viewport_height");

#ifdef PROPER_CELL
        glpl_.program->set_uniform(base_level_resolution_, "base_level_resolution");
        glpl_.program->set_uniform(1, "shading_type");
#endif

        glpl_.vao->draw(GL_POINTS, 0, static_cast<int>(interop_cells_->size()));
    });
}

void RenderableDvh::resize(const num::ivec2& viewport) {
    viewport_height_ = viewport.y;
}

void RenderableDvh::set_camera_pos(num::vec3 cam_pos) {
    camera_position_ = cam_pos;
}
