#include "hip/hip_runtime.h"
// ///////////////////////////////////////////////////////////////////////////////////////
// LTB Distance Volume Hierarchy
// Copyright (c) 2020 Logan Barnes - All Rights Reserved
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
// ///////////////////////////////////////////////////////////////////////////////////////
#include "dvh_renderable.hpp"

// project
#include "buffer_map_guard.hpp"

// external
#include <Magnum/Math/Vector2.h>
#include <glm/ext/scalar_constants.hpp>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

// standard
#include <algorithm>
#include <random>

namespace ltb {
namespace example {

namespace {

using namespace Magnum::Math::Literals;

struct DistanceFromCameraComparator {
    LTB_CUDA_FUNC bool operator()(const Cell& c1, const Cell& c2) {
        return glm::distance(camera_position, c1.center_point(level_0_resolution))
            > glm::distance(camera_position, c2.center_point(level_0_resolution));
    }

    glm::vec3 camera_position;
    float     level_0_resolution;
};

auto create_cells() {
    constexpr auto index_radius = 20.f;
    constexpr auto num_points   = 1000000u;

    // Create a bunch of uniformly distributed points on a sphere
    std::vector<glm::vec3> points;
    points.reserve(num_points);
    {
        float                                 u, theta, coeff;
        std::mt19937                          gen{std::random_device{}()};
        std::uniform_real_distribution<float> u_dist(-1.f, 1.f);
        std::uniform_real_distribution<float> theta_dist(0.f, 2.f * glm::pi<float>());

        for (auto i = 0u; i < num_points; ++i) {
            // Uniform surface distribution
            u     = u_dist(gen);
            theta = theta_dist(gen);
            coeff = std::sqrt(1.f - u * u);

            points.emplace_back(coeff * std::cos(theta), coeff * std::sin(theta), u);
        }
    }

    std::vector<Cell> cells(points.size());

    std::transform(points.begin(), points.end(), cells.begin(), [](const auto& p3) {
        return Cell{glm::ivec3(glm::round(p3 * index_radius)), glm::normalize(p3), 3};
    });

    return cells;
}

} // namespace

LTB_CUDA_FUNC auto Cell::center_point(float level_0_resolution) const -> glm::vec3 {
    return glm::vec3(index) * level_0_resolution * glm::pow(2.f, static_cast<float>(level));
}

DvhRenderable::DvhRenderable(glm::ivec2 viewport) {
    {
        auto cells     = create_cells();
        interop_cells_ = std::make_unique<cuda::GLBuffer<Cell>>(cells);
    }

    glEnable(GL_PROGRAM_POINT_SIZE);

    mesh_.addVertexBuffer(interop_cells_->gl_buffer(),
                          0,
                          dvh::CellShader::Index(),
                          dvh::CellShader::VectorToClosestPoint(),
                          dvh::CellShader::Level());
    mesh_.setCount(interop_cells_->size());
    mesh_.setPrimitive(Magnum::GL::MeshPrimitive::Points);

    resize(viewport);
}

void DvhRenderable::update(double /*time_step*/) {
    auto guard = cuda::make_gl_buffer_map_guard(*interop_cells_);

    std::size_t buffer_size;
    auto*       raw_device_ptr = interop_cells_->cuda_buffer(&buffer_size);

    auto device_ptr = thrust::device_pointer_cast(raw_device_ptr);

    thrust::sort(device_ptr,
                 device_ptr + static_cast<decltype(device_ptr)::difference_type>(buffer_size),
                 DistanceFromCameraComparator{camera_position_, base_level_resolution_});
}

void DvhRenderable::render(const gvs::CameraPackage& camera_package) const {

    glEnable(GL_BLEND);
    glDepthMask(GL_FALSE);

    glBlendEquationSeparate(GL_FUNC_ADD, GL_FUNC_ADD);
    glBlendFuncSeparate(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA, GL_ONE, GL_ZERO);

    shader_
        .set_projection_from_world_matrix(camera_package.camera->projectionMatrix()
                                          * camera_package.camera->cameraMatrix())
        .set_projection_from_view_matrix(camera_package.camera->projectionMatrix())
        .set_viewport_height(viewport_.y)
        .set_base_level_resolution(base_level_resolution_)
        .set_coloring_type(dvh::CellColoring::Normals)
        .set_shading_type(gvs::Shading::UniformColor);

    mesh_.draw(shader_);

    glDepthMask(GL_TRUE);
    glDisable(GL_BLEND);
}

void DvhRenderable::configure_gui() {}

void DvhRenderable::resize(glm::ivec2 viewport) {
    viewport_ = viewport;
}

} // namespace example
} // namespace ltb
