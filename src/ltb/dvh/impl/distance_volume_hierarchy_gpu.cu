#include "hip/hip_runtime.h"
// ///////////////////////////////////////////////////////////////////////////////////////
// LTB Distance Volume Hierarchy
// Copyright (c) 2020 Logan Barnes - All Rights Reserved
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
// ///////////////////////////////////////////////////////////////////////////////////////
#include "distance_volume_hierarchy_gpu.hpp"

// external
#include <doctest/doctest.h>

namespace ltb {
namespace dvh {

template <int L, typename T>
DistanceVolumeHierarchyGpu<L, T>::DistanceVolumeHierarchyGpu(T base_resolution, int max_level)
    : base_resolution_(base_resolution), max_level_(max_level) {
    clear();
}

template <int L, typename T>
void DistanceVolumeHierarchyGpu<L, T>::clear() {
    levels_.clear();
}

template <int L, typename T>
void DistanceVolumeHierarchyGpu<L, T>::actually_add_volume(std::vector<sdf::Geometry<L, T> const*> const& geometries) {
    if (geometries.empty()) {
        return;
    }

    auto volume_bounds = sdf::AABB<L, T>();

    for (auto const* geometry : geometries) {
        auto aabb     = geometry->bounding_box();
        volume_bounds = sdf::expand(volume_bounds, aabb.min_point);
        volume_bounds = sdf::expand(volume_bounds, aabb.max_point);
    }

    add_roots_for_bounds(volume_bounds);

    // ///////////////////////////////////////////////// //

    CellSet to_visit;
    CellSet cells;

    for (int level = roots_.begin()->first; level >= lowest_level_; --level) {

        cells = std::move(to_visit);
        to_visit.clear(); // Just to make sure

        if (roots_.find(level) != roots_.end()) {
            auto const& root_cells = roots_.at(level);
            cells.insert(root_cells.begin(), root_cells.end());
        }

        auto level_resolution = resolution(level);
        auto half_resolution  = level_resolution * T(0.5);
        auto cell_corner_dist = glm::length(glm::vec<L, T>(half_resolution));

        auto& distance_field = levels_[level];

        for (const auto& cell : cells) {
            auto const p = dvh::cell_center(cell, level_resolution);

            auto min_dist     = std::numeric_limits<T>::infinity();
            auto min_abs_dist = min_dist;

            for (auto const* geometry : geometries) {
                auto const dist     = geometry->distance_from(p);
                auto const abs_dist = std::abs(dist);

                if (should_replace_with(min_abs_dist, abs_dist, dist)) {
                    min_dist     = dist;
                    min_abs_dist = abs_dist;
                }
            }

            // TODO: double check this logic for already existing cells with smaller distances
            // (make sure children are still visited if necessary)
            if (distance_field.find(cell) == distance_field.end()
                || should_replace_with(std::abs(distance_field.at(cell)[L]), min_abs_dist, min_dist)) {
                if (min_dist <= cell_corner_dist) {
                    distance_field[cell] = glm::vec<L + 1, T>(p, min_dist);

                    if (min_abs_dist <= cell_corner_dist && level > lowest_level_) {
                        auto                                                     children = children_cells(cell);
                        typedef typename std::vector<glm::vec<L, int>>::iterator iter_t;
                        to_visit.insert(std::move_iterator<iter_t>(children.begin()),
                                        std::move_iterator<iter_t>(children.end()));
                    }
                }
            }
        }
    }
}

template <int L, typename T>
auto DistanceVolumeHierarchyGpu<L, T>::levels() const -> LevelMap<SparseVolumeMap> const& {
    return levels_;
}

template <int L, typename T>
auto DistanceVolumeHierarchyGpu<L, T>::base_resolution() const -> T {
    return base_resolution_;
}

template <int L, typename T>
auto DistanceVolumeHierarchyGpu<L, T>::resolution(int level_index) const -> T {
    return base_resolution_ * std::pow<T>(2, level_index);
}

template <int L, typename T>
auto DistanceVolumeHierarchyGpu<L, T>::add_roots_for_bounds(const sdf::AABB<L, T>& aabb) -> void {

    auto root_level = lowest_level_;
    auto min_cell   = glm::vec<L, int>();
    auto max_cell   = glm::vec<L, int>();
    auto dimensions = glm::vec<L, int>(std::numeric_limits<int>::max());

    for (int level = root_level; level < max_level_; ++level) {
        auto level_resolution = resolution(level);

        auto level_min_cell = get_cell(aabb.min_point, level_resolution);
        auto level_max_cell = get_cell(aabb.max_point, level_resolution);

        auto level_dimensions = level_max_cell - level_min_cell;

        if (level_dimensions == dimensions) {
            break;
        }

        root_level = level;
        min_cell   = level_min_cell;
        max_cell   = level_max_cell;
        dimensions = level_dimensions;
    }

    auto& roots = roots_[root_level];

    iterate(min_cell, max_cell, [&roots](auto const& cell) { roots.emplace(cell); });
}

template class DistanceVolumeHierarchyGpu<2, float>;
template class DistanceVolumeHierarchyGpu<3, float>;
template class DistanceVolumeHierarchyGpu<2, double>;
template class DistanceVolumeHierarchyGpu<3, double>;

} // namespace dvh
} // namespace ltb
