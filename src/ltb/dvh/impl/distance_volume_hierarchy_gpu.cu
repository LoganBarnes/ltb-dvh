#include "hip/hip_runtime.h"
// ///////////////////////////////////////////////////////////////////////////////////////
// LTB Distance Volume Hierarchy
// Copyright (c) 2020 Logan Barnes - All Rights Reserved
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
// ///////////////////////////////////////////////////////////////////////////////////////
#include "distance_volume_hierarchy_gpu.hpp"

// project
#include "ltb/dvh/distance_volume_hierarchy_util.hpp"
#include "ltb/sdf/sdf.hpp"

// external
#include <doctest/doctest.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

//namespace ltb {
//namespace dvh {
//
//template <int L, typename T>
//struct VolumeCell {
//    Cell   index;
//    glm::vec<L + 1, T> direction_and_distance = glm::vec<L + 1, T>(std::numeric_limits<T>::infinity());
//
//    explicit VolumeCell(Cell cell) : index(cell) {}
//
//    explicit VolumeCell(Cell cell, glm::vec<L + 1, T> dir_and_dist)
//        : index(cell), direction_and_distance(dir_and_dist) {}
//};
//
//template <int L, typename T>
//LTB_CUDA_FUNC auto operator==(VolumeCell<L, T> const& lhs, VolumeCell<L, T> const& rhs) -> bool {
//    return lhs.index == rhs.index;
//}
//
//} // namespace dvh
//} // namespace ltb
//
//namespace std {
//
//template <int L, typename T>
//struct hash<ltb::dvh::VolumeCell<L, T>> {
//    size_t operator()(ltb::dvh::VolumeCell<L, T> const& cell) const { return hash<Cell>{}(cell.index); }
//};
//
//} // namespace std

namespace ltb {
namespace dvh {

template <int L, typename T>
DistanceVolumeHierarchyGpu<L, T>::DistanceVolumeHierarchyGpu(T base_resolution, int max_level)
    : base_resolution_(base_resolution), max_level_(max_level) {
    clear();
}

template <int L, typename T>
void DistanceVolumeHierarchyGpu<L, T>::clear() {
    cpu_levels_.clear();
}

template <int L, typename T>
auto DistanceVolumeHierarchyGpu<L, T>::levels() const -> LevelMap<SparseVolumeMap> const& {
    return cpu_levels_;
}

template <int L, typename T>
auto DistanceVolumeHierarchyGpu<L, T>::base_resolution() const -> T {
    return base_resolution_;
}

template <int L, typename T>
auto DistanceVolumeHierarchyGpu<L, T>::resolution(int level_index) const -> T {
    return base_resolution_ * std::pow<T>(2, level_index);
}

template <int L, typename T>
auto DistanceVolumeHierarchyGpu<L, T>::add_roots_for_bounds(const sdf::AABB<L, T>& aabb) -> void {

    auto root_level = lowest_level_;
    auto min_cell   = Cell();
    auto max_cell   = Cell();
    auto dimensions = Cell(std::numeric_limits<int>::max());

    for (int level = root_level; level < max_level_; ++level) {
        auto level_resolution = resolution(level);

        auto level_min_cell = get_cell(aabb.min_point, level_resolution);
        auto level_max_cell = get_cell(aabb.max_point, level_resolution);

        auto level_dimensions = level_max_cell - level_min_cell;

        if (level_dimensions == dimensions) {
            break;
        }

        root_level = level;
        min_cell   = level_min_cell;
        max_cell   = level_max_cell;
        dimensions = level_dimensions;
    }

    auto& roots = cpu_roots_[root_level];

    iterate(min_cell, max_cell, [&roots](auto const& cell) { roots.emplace(cell); });
}

template class DistanceVolumeHierarchyGpu<2, float>;
template class DistanceVolumeHierarchyGpu<3, float>;
template class DistanceVolumeHierarchyGpu<2, double>;
template class DistanceVolumeHierarchyGpu<3, double>;

} // namespace dvh
} // namespace ltb
