#include "hip/hip_runtime.h"
// ///////////////////////////////////////////////////////////////////////////////////////
// LTB Distance Volume Hierarchy
// Copyright (c) 2020 Logan Barnes - All Rights Reserved
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
// ///////////////////////////////////////////////////////////////////////////////////////
#include "dvh_renderable.hpp"

// project
#include "buffer_map_guard.hpp"

// standard
#include <random>

namespace ltb {
namespace example {

namespace {

//// Cuda kernel to test the CUDA side of things
//__global__ void update_particles(gpu::device_buffer<Particle> particles, float timestep) {
//    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if (idx >= particles.length()) {
//        return;
//    }
//
//    // Apply euler update
//    auto& particle = particles[idx];
//    particle.position += particle.velocity * timestep;
//
//    // reverse velocity ("bounce") if we hit a 100x100x100 cube
//    for (int i = 0; i < 3; ++i) {
//        if (fabsf(particle.position[i]) > 50.f) {
//            particle.velocity[i] = -particle.velocity[i];
//        }
//    }
//}

} // namespace

DvhRenderable::DvhRenderable() {
    // create 1000000 random particles within a 70x70x70 cube
    {
        std::mt19937                          gen(std::random_device{}());
        std::uniform_real_distribution<float> dist(-35.f, 35.f);
        std::vector<Particle>                 particles(1000000);

        for (auto& particle : particles) {
            particle = {{dist(gen), dist(gen), dist(gen)}, glm::normalize(glm::vec3(dist(gen), dist(gen), dist(gen)))};
        }

        interop_particles_ = std::make_unique<cuda::GLBuffer<Particle>>(particles);
    }

    glEnable(GL_PROGRAM_POINT_SIZE);
    //    glpl_.program = std::make_shared<gl::Program>(gl::points_vert(), gl::points_frag());
    //    glpl_.vao     = std::make_shared<gl::VertexArray>(*glpl_.program,
    //                                                  interop_particles_->gl_buffer(),
    //                                                  sizeof(Particle),
    //                                                  gl::pos_float_vao_elements());
}

void DvhRenderable::update(double time_step) {
    auto guard = cuda::make_gl_buffer_map_guard(*interop_particles_);
    //    gpu::launch_default(update_particles,
    //                        interop_particles_->size(),
    //                        *interop_particles_,
    //                        static_cast<float>(time_step));
}

void DvhRenderable::render(const gvs::CameraPackage& /*camera_package*/) const {
    // OpenGL rendering to test the OpenGL side of things
    //    glpl_.program->use([&] {
    //        glpl_.program->set_uniform(camera.get_projection_view_matrix(), "world_from_local");
    //        glpl_.program->set_uniform(viewport_height, "view_height");
    //        glpl_.program->set_uniform(camera.get_projection_matrix(), "projection_from_view");
    //        glpl_.program->set_uniform(global_particle_radius, "point_radius");
    //        glpl_.program->set_uniform(camera.get_view_matrix(), "view_from_world");
    //
    //        glpl_.vao->draw(GL_POINTS, 0, static_cast<int>(interop_particles_->size()));
    //    });
}

void DvhRenderable::configure_gui() {}

} // namespace example
} // namespace ltb
